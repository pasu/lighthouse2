#include "hip/hip_runtime.h"
/* camera.cu - Copyright 2019 Utrecht University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include "noerrors.h"

//  +-----------------------------------------------------------------------------+
//  |  generateEyeRaysKernel                                                      |
//  |  Generate primary rays, to be traced by Optix Prime.                  LH2'19|
//  +-----------------------------------------------------------------------------+
__global__  __launch_bounds__( 256 , 1 )
void constructionLightPosKernel(int smcount, float NKK,uint* constructLightBuffer, 
    BiPathState* pathStateData, const uint R0, const uint* blueNoise, const int4 screenParams,
    Ray4* randomWalkRays)
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= counters->activePaths) return;

    int jobIndex = constructLightBuffer[gid];

    const int scrhsize = screenParams.x & 0xffff;
    const int scrvsize = screenParams.x >> 16;
    const uint x = jobIndex % scrhsize;
    uint y = jobIndex / scrhsize;

    uint path_s_t_type_pass = pathStateData[jobIndex].pathInfo.w;

    uint s = 0;
    uint t = 1;
    uint type = 0;
    uint sampleIdx = path_s_t_type_pass & 255;

    float r0,r1,r2,r3;

    if (sampleIdx < 256)
    {
        r0 = blueNoiseSampler(blueNoise, x, y, sampleIdx, 0);
        r1 = blueNoiseSampler(blueNoise, x, y, sampleIdx, 1);
        r2 = blueNoiseSampler(blueNoise, x, y, sampleIdx, 2);
        r3 = blueNoiseSampler(blueNoise, x, y, sampleIdx, 3);
    }
    else
    {
        uint seed = WangHash(jobIndex + R0);

        r0 = RandomFloat(seed);
        r1 = RandomFloat(seed);
        r2 = RandomFloat(seed);
        r3 = RandomFloat(seed);
    }

    float3 normal, throughput, lightDir;
    float lightPdf, pdfPos, pdfDir ;

    float3 pos = Sample_Le(r0, r1, r2, r3, normal, lightDir, throughput, lightPdf, pdfPos, pdfDir);

    float3 beta = throughput * dot(normal, lightDir) / (lightPdf * pdfPos * pdfDir);

    float light_p = lightPdf * pdfPos;
    float dL = NKK / light_p;
    float light_pdf_solid = pdfDir;

    const uint randomWalkRayIdx = atomicAdd(&counters->randomWalkRays, 1);
    randomWalkRays[randomWalkRayIdx].O4 = make_float4(pos, EPSILON);
    randomWalkRays[randomWalkRayIdx].D4 = make_float4(lightDir, 1e34f);

    pathStateData[jobIndex].data0 = make_float4(throughput, dL);
    pathStateData[jobIndex].data1 = make_float4(beta, light_p);
    pathStateData[jobIndex].data2 = make_float4(pos, light_pdf_solid);
    pathStateData[jobIndex].data3 = make_float4(lightDir, __int_as_float(randomWalkRayIdx));
    pathStateData[jobIndex].light_normal = make_float4(normal, 0.0f);

    sampleIdx++;
    path_s_t_type_pass = (s << 24) + (t<<16) + (type<<8) + sampleIdx;

    pathStateData[jobIndex].pathInfo.w = path_s_t_type_pass;
}

//  +-----------------------------------------------------------------------------+
//  |  constructionLightPos                                                            |
//  |  Entry point for the persistent constructionLightPos kernel.               LH2'19|
//  +-----------------------------------------------------------------------------+
__host__ void constructionLightPos( int smcount, float NKK, uint* constructLightBuffer, 
    BiPathState* pathStateData, const uint R0, const uint* blueNoise, const int4 screenParams,
    Ray4* randomWalkRays)
{
	const dim3 gridDim( NEXTMULTIPLEOF(smcount, 256 ) / 256, 1 ), blockDim( 256, 1 );
    constructionLightPosKernel << < gridDim.x, 256 >> > (smcount, NKK, constructLightBuffer, pathStateData, R0, blueNoise, screenParams, randomWalkRays);
}

// EOF