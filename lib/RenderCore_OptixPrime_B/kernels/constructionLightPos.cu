#include "hip/hip_runtime.h"
/* camera.cu - Copyright 2019 Utrecht University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include "noerrors.h"

//  +-----------------------------------------------------------------------------+
//  |  generateEyeRaysKernel                                                      |
//  |  Generate primary rays, to be traced by Optix Prime.                  LH2'19|
//  +-----------------------------------------------------------------------------+
__global__  __launch_bounds__( 256 , 1 )
void constructionLightPosKernel(int smcount, float NKK,uint* constructLightBuffer, 
    BiPathState* pathStateData, const uint R0, const uint* blueNoise, const int4 screenParams,
    Ray4* randomWalkRays, float4* accumulatorOnePass, float4* accumulator,
    float4* weightMeasureBuffer, const int probePixelIdx)
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= counters->activePaths) return;

    int jobIndex = constructLightBuffer[gid];

    const int scrhsize = screenParams.x & 0xffff;
    const int scrvsize = screenParams.x >> 16;
    const uint x = jobIndex % scrhsize;
    uint y = jobIndex / scrhsize;

    uint path_s_t_type_pass = pathStateData[jobIndex].pathInfo.w;

    uint s = 0;
    uint t = 1;
    uint type = 0;
    uint sampleIdx = path_s_t_type_pass & 524287;//2^19-1
    /*
    if (jobIndex == probePixelIdx)
    {
        uint pass, eye, light, c;

        getPathInfo(path_s_t_type_pass,pass,eye,light,c);
        //printf("%d,%d\n", eye,light);
        //float4 v4 = weightMeasureBuffer[jobIndex];
        //float fSum = v4.x + v4.y + v4.z + v4.w;
        //printf("%f,%f,%f,%f\n", v4.x / fSum, v4.y / fSum, v4.z / fSum, v4.w / fSum);

        float4 color = accumulatorOnePass[jobIndex];
        printf("%f,%f,%f,%d,%d,%d\n", color.x, color.y, color.z, eye,light, sampleIdx);
    }
    */
    accumulator[jobIndex] += accumulatorOnePass[jobIndex];
    accumulator[jobIndex].w = sampleIdx;
    accumulatorOnePass[jobIndex] = make_float4(0.0f);
    weightMeasureBuffer[jobIndex] = make_float4(0.0f);

    float r0,r1,r2,r3;

    if (sampleIdx < 256)
    {
        r0 = blueNoiseSampler(blueNoise, x, y, sampleIdx, 0);
        r1 = blueNoiseSampler(blueNoise, x, y, sampleIdx, 1);
        r2 = blueNoiseSampler(blueNoise, x, y, sampleIdx, 2);
        r3 = blueNoiseSampler(blueNoise, x, y, sampleIdx, 3);
    }
    else
    {
        uint seed = WangHash(jobIndex + R0);

        r0 = RandomFloat(seed);
        r1 = RandomFloat(seed);
        r2 = RandomFloat(seed);
        r3 = RandomFloat(seed);
    }

    float3 normal, throughput, lightDir;
    float lightPdf, pdfPos, pdfDir ;

    float3 pos = Sample_Le(r0, r1, r2, r3, normal, lightDir, throughput, lightPdf, pdfPos, pdfDir);

    // PBR book equation [16.15]
    float3 beta = throughput * fabs(dot(normal, lightDir)) / (lightPdf * pdfPos * pdfDir);

    float light_p = lightPdf * pdfPos;
    float dL = NKK / light_p;
    float light_pdf_solid = pdfDir;

    const uint randomWalkRayIdx = atomicAdd(&counters->randomWalkRays, 1);
    randomWalkRays[randomWalkRayIdx].O4 = make_float4(SafeOrigin(pos, lightDir, normal, geometryEpsilon), 0);
    randomWalkRays[randomWalkRayIdx].D4 = make_float4(lightDir, 1e34f);

    pathStateData[jobIndex].data0 = make_float4(throughput, dL);
    pathStateData[jobIndex].data1 = make_float4(beta, light_p);
    pathStateData[jobIndex].data2 = make_float4(pos, light_pdf_solid);
    pathStateData[jobIndex].data3 = make_float4(lightDir, __int_as_float(randomWalkRayIdx));
    pathStateData[jobIndex].light_normal = make_float4(normal, 0.0f);

    sampleIdx++;
    path_s_t_type_pass = (s << 27) + (t<<22) + (type<<19) + sampleIdx;

    pathStateData[jobIndex].pathInfo.w = path_s_t_type_pass;
}

//  +-----------------------------------------------------------------------------+
//  |  constructionLightPos                                                            |
//  |  Entry point for the persistent constructionLightPos kernel.               LH2'19|
//  +-----------------------------------------------------------------------------+
__host__ void constructionLightPos( int smcount, float NKK, uint* constructLightBuffer, 
    BiPathState* pathStateData, const uint R0, const uint* blueNoise, const int4 screenParams,
    Ray4* randomWalkRays, float4* accumulatorOnePass, float4* accumulator,
    float4* weightMeasureBuffer, const int probePixelIdx)
{
	const dim3 gridDim( NEXTMULTIPLEOF(smcount, 256 ) / 256, 1 ), blockDim( 256, 1 );
    constructionLightPosKernel << < gridDim.x, 256 >> > (smcount, NKK, constructLightBuffer, 
        pathStateData, R0, blueNoise, screenParams, randomWalkRays,
        accumulatorOnePass, accumulator, weightMeasureBuffer,probePixelIdx);
}

// EOF