#include "hip/hip_runtime.h"
/* camera.cu - Copyright 2019 Utrecht University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include "noerrors.h"

//  +-----------------------------------------------------------------------------+
//  |  constructionLightPosKernel                                                      |
//  |  Generate the first vertex of the light path including pos and direction.                  LH2'19|
//  +-----------------------------------------------------------------------------+
__global__  __launch_bounds__( 256 , 1 )
void finalizeConnectionsKernel(int smcount, float4* accumulatorOnePass, 
    float4* accumulator)
{
    int jobIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (jobIndex >= smcount) return;

    accumulator[jobIndex] += accumulatorOnePass[jobIndex];
    accumulator[jobIndex].w += 1.0f;
}

//  +-----------------------------------------------------------------------------+
//  |  constructionLightPos                                                            |
//  |  Entry point for the persistent constructionLightPos kernel.               LH2'19|
//  +-----------------------------------------------------------------------------+
__host__ void finalizeConnections( int smcount, float4* accumulatorOnePass, 
    float4* accumulator)
{
	const dim3 gridDim( NEXTMULTIPLEOF(smcount, 256 ) / 256, 1 ), blockDim( 256, 1 );
    finalizeConnectionsKernel << < gridDim.x, 256 >> > (smcount,
        accumulatorOnePass, accumulator);
}

// EOF