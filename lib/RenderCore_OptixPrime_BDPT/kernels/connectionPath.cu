#include "hip/hip_runtime.h"
/* camera.cu - Copyright 2019 Utrecht University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include "noerrors.h"

#define INSTANCEIDX (prim >> 20)
#define HIT_U hitData.x
#define HIT_V hitData.y
#define HIT_T hitData.w

LH2_DEVFUNC void Sample_Wi(const float aperture, const float imgPlaneSize, const float3 eye_pos,
    const float3 forward, const float3 light_pos, const float focalDistance,
    const float3 p1, const float3 right, const float3 up,
    float3& throughput, float& pdf, float& u, float& v)
{
    throughput = make_float3(0.0f);
    pdf = 0.0f;

    float3 dir = light_pos - eye_pos;
    float dist = length(dir);

    dir /= dist;

    float cosTheta = dot(normalize(forward), dir);

    // check direction
    if (cosTheta <= 0)
    {
        return;
    }

    float x_length = length(right);
    float y_length = length(up);

    float distance = focalDistance / cosTheta;

    float3 raster_pos = eye_pos + distance * dir;
    float3 pos2p1 = raster_pos - p1;

    float3 unit_up = up / y_length;
    float3 unit_right = right / x_length;

    float x_offset = dot(unit_right, pos2p1);
    float y_offset = dot(unit_up, pos2p1);

    // check view fov
    if (x_offset<0 || x_offset > x_length
        || y_offset<0 || y_offset > y_length)
    {
        //printf("%f,%f,%f,%f\n", x_offset, x_length,y_offset, y_length);
        return;
    }

    //printf("in raster\n");

    u = x_offset / x_length;
    v = y_offset / y_length;

    float cos2Theta = cosTheta * cosTheta;
    float lensArea = aperture != 0 ? aperture * aperture * PI : 1;
    lensArea = 1.0f; // because We / pdf
    float We = 1.0f / (imgPlaneSize * lensArea * cos2Theta * cos2Theta);

    throughput = make_float3(We);
    pdf = dist * dist / (cosTheta * lensArea);
}

//  +-----------------------------------------------------------------------------+
//  |  generateEyeRaysKernel                                                      |
//  |  Generate primary rays, to be traced by Optix Prime.                  LH2'19|
//  +-----------------------------------------------------------------------------+
__global__  __launch_bounds__( 256 , 1 )
void connectionPathKernel(int smcount, float NKK, float scene_area, BiPathState* pathStateData,
    const Intersection* randomWalkHitBuffer, uint* visibilityHitBuffer,
    const float aperture, const float imgPlaneSize, const float3 forward,
    const float focalDistance, const float3 p1, const float3 right, const float3 up,
    const float spreadAngle, float4* accumulatorOnePass, float4* accumulator, uint* constructLightBuffer,
    float4* weightMeasureBuffer, const int probePixelIdx, const int4 screenParams,
    uint* photomappingIdx, float4* photomappingBuffer, const float3 camPos,
    uint* constructEyeBuffer, uint* eyePathBuffer, uint* lightPathBuffer)
{
    int jobIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (jobIndex >= smcount) return;

    const int scrhsize = screenParams.x & 0xffff;
    const int scrvsize = screenParams.x >> 16;

    const uint x_line = jobIndex % scrhsize;
    uint y_line = jobIndex / scrhsize;

    uint path_s_t_type_pass = pathStateData[jobIndex].pathInfo.w;

    uint pass, type, t, s;
    getPathInfo(path_s_t_type_pass,pass,s,t,type);

    const float3 empty_color = make_float3(0.0f);
    float3 L = empty_color;
    float misWeight = 0.0f;

    const uint occluded = visibilityHitBuffer[jobIndex >> 5] & (1 << (jobIndex & 31));

    bool bAddImplicitPath = true;
    bool bAddExplicitPath = true;
    bool bAddCombinedPath = true;
    bool bAddPhotoMappingPath = true;

    if (type == 2 && bAddPhotoMappingPath)
    {
        float3 light_pos = make_float3(pathStateData[jobIndex].data2);
        float3 eye_pos = camPos;

        float3 light2eye = eye_pos - light_pos;
        float length_l2e = length(light2eye);
        light2eye /= length_l2e;

        float3 throughput_eye;
        float pdf_eye;
        float u, v;
        Sample_Wi(aperture,imgPlaneSize,eye_pos,forward,light_pos,
            focalDistance, p1, right, up, throughput_eye, pdf_eye, u, v);

        if (pdf_eye > EPSILON)
        {
            float4 hitData = pathStateData[jobIndex].currentLight_hitData;
            float3 dir = make_float3(pathStateData[jobIndex].pre_light_dir);

            const int prim = __float_as_int(hitData.z);
            const int primIdx = prim == -1 ? prim : (prim & 0xfffff);

            const CoreTri4* instanceTriangles = (const CoreTri4*)instanceDescriptors[INSTANCEIDX].triangles;

            ShadingData shadingData;
            float3 N, iN, fN, T;

            const float coneWidth = spreadAngle * HIT_T;
            GetShadingData(dir, HIT_U, HIT_V, coneWidth, instanceTriangles[primIdx], INSTANCEIDX, shadingData, N, iN, fN, T);

            float bsdfPdf;
            const float3 sampledBSDF = EvaluateBSDF(shadingData, fN, T, dir * -1.0f, light2eye, bsdfPdf);

            float3 light_throught = make_float3(pathStateData[jobIndex].data0);
            float cosTheta = fabs(dot(fN, light2eye));

            float eye_cosTheta = fabs(dot(normalize(forward), light2eye * -1.0f));
            float eye_pdf_solid = 1.0f / (imgPlaneSize * eye_cosTheta * eye_cosTheta * eye_cosTheta);
            float p_forward = eye_pdf_solid * cosTheta / (length_l2e * length_l2e);

            float dL = pathStateData[jobIndex].data0.w;

            misWeight = 1.0f / (1 + dL * p_forward);

            if (!occluded)
            {
                uint x = (scrhsize * u + 0.5);
                uint y = (scrvsize * v + 0.5);
                uint idx = y * scrhsize + x;

                L = light_throught * sampledBSDF * (throughput_eye / pdf_eye) * cosTheta;

                /*
                float pdf_solidangle = pathStateData[jobIndex].data2.w;
                if (fabs(pdf_solidangle - 1.0f) < EPSILON)
                {
                    L = empty_color;
                }
                */

                //misWeight = 1.0f;
                float4 res_color = make_float4((L*misWeight), misWeight);
                atomicAdd(&(accumulatorOnePass[idx].x), res_color.x);
                atomicAdd(&(accumulatorOnePass[idx].y), res_color.y);
                atomicAdd(&(accumulatorOnePass[idx].z), res_color.z);
                atomicAdd(&(accumulatorOnePass[idx].w), res_color.w);
                //accumulatorOnePass[idx] += ;
                //weightMeasureBuffer[idx].w += misWeight;

                const uint pm_idx = atomicAdd(&counters->photomappings, 1);

                photomappingBuffer[pm_idx] = make_float4(L, __int_as_float(idx));

                /*
                if (idx == probePixelIdx)
                {
                    printf("Photon:%f,%f,%f,%f,%d,%d\n", L.x, L.y, L.z, misWeight, s, t);
                }
                */

                L = make_float3(0.0f);
                //misWeight = 0.0f;
                
            }
            //printf("w:%f\n", misWeight);
            /*
            if (bsdfPdf < EPSILON || isnan(bsdfPdf))
            {
                L = empty_color;
            }
            */
        }
    }
    //misWeight = 1.0f;
    accumulatorOnePass[jobIndex] += make_float4((L*misWeight), misWeight);
    
    int eye_hit = -1;
    int eye_hit_idx = __float_as_int(pathStateData[jobIndex].data7.w);
    float eye_pdf = pathStateData[jobIndex].data6.w;
    if (eye_pdf < EPSILON || isnan(eye_pdf))
    {
        eye_hit = -1;
        pathStateData[jobIndex].data7.w = __int_as_float(-1);
    }
    else if (eye_hit_idx > -1)
    {
        const Intersection hd = randomWalkHitBuffer[eye_hit_idx];

        eye_hit = hd.triid;

        const float4 hitData = make_float4(hd.u, hd.v, __int_as_float(hd.triid + (hd.triid == -1 ? 0 : (hd.instid << 20))), hd.t);
        pathStateData[jobIndex].eye_intersection = hitData;

        pathStateData[jobIndex].data7.w = __int_as_float(-1);
    }

    int light_hit = -1;
    int light_hit_idx = __float_as_int(pathStateData[jobIndex].data3.w);
    float light_pdf_test = pathStateData[jobIndex].data2.w;
    if (light_pdf_test < EPSILON || isnan(light_pdf_test))
    {
        light_hit = -1;
        pathStateData[jobIndex].data3.w = __int_as_float(-1);
    }
    else if (light_hit_idx > -1)
    {
        const Intersection hd = randomWalkHitBuffer[light_hit_idx];
        light_hit = hd.triid;
        const float4 hitData = make_float4(hd.u, hd.v, __int_as_float(hd.triid + (hd.triid == -1 ? 0 : (hd.instid << 20))), hd.t);

        pathStateData[jobIndex].light_intersection = hitData;
        pathStateData[jobIndex].data3.w = __int_as_float(-1);
    }
    else
    {
        const float4 hitData = pathStateData[jobIndex].light_intersection;

        const int prim = __float_as_int(hitData.z);
        const int primIdx = prim == -1 ? prim : (prim & 0xfffff);

        light_hit = primIdx;
    }

    const uint MAX__LENGTH_E = 3;
    const uint MAX__LENGTH_L = 5;

    if (eye_hit != -1 && s < MAX__LENGTH_E)
    {
        type = 1;
        const uint eyePIdx = atomicAdd(&counters->extendEyePath, 1);
        eyePathBuffer[eyePIdx] = jobIndex;
    }
    else if (light_hit != -1 && t < MAX__LENGTH_L)
    {
        type = 2;

        const uint eyeIdx = atomicAdd(&counters->constructionEyePos, 1);
        constructEyeBuffer[eyeIdx] = jobIndex;

        const uint lightPIdx = atomicAdd(&counters->extendLightPath, 1);
        lightPathBuffer[lightPIdx] = jobIndex;
    }
    else
    {
        const uint constructLight = atomicAdd(&counters->constructionLightPos, 1);
        constructLightBuffer[constructLight] = jobIndex;
    }

    if (eye_hit == -1 && type != 2)
    {
        float3 hit_dir = make_float3(pathStateData[jobIndex].data7);
        float3 background = make_float3(SampleSkydome(hit_dir, s+1));

        // hit miss : beta 
        float3 beta = make_float3(pathStateData[jobIndex].data5);
        float3 contribution = beta * background;

        CLAMPINTENSITY; // limit magnitude of thoughput vector to combat fireflies
        FIXNAN_FLOAT3(contribution);

        float dE = pathStateData[jobIndex].data4.w;
        misWeight = 1.0f;// / (dE * (1.0f / (scene_area)) + NKK);

        accumulatorOnePass[jobIndex] += make_float4((contribution * misWeight), misWeight);
    }

    path_s_t_type_pass = (s << 27) + (t << 22) + (type << 19) + pass;
    pathStateData[jobIndex].pathInfo.w = path_s_t_type_pass;
}

//  +-----------------------------------------------------------------------------+
//  |  constructionLightPos                                                            |
//  |  Entry point for the persistent constructionLightPos kernel.               LH2'19|
//  +-----------------------------------------------------------------------------+
__host__ void connectionPath(int smcount, float NKK, float scene_area, BiPathState* pathStateBuffer,
    const Intersection* randomWalkHitBuffer, uint* visibilityHitBuffer,
    const float aperture, const float imgPlaneSize, const float3 forward, 
    const float focalDistance, const float3 p1, const float3 right, const float3 up,
    const float spreadAngle, float4* accumulatorOnePass, float4* accumulator, uint* constructLightBuffer,
    float4* weightMeasureBuffer, const int probePixelIdx, const int4 screenParams,
    uint* photomappingIdx, float4* photomappingBuffer, const float3 camPos, 
    uint* constructEyeBuffer, uint* eyePathBuffer, uint* lightPathBuffer)
{
	const dim3 gridDim( NEXTMULTIPLEOF(smcount, 256 ) / 256, 1 ), blockDim( 256, 1 );
    connectionPathKernel << < gridDim.x, 256 >> > (smcount, NKK, scene_area, pathStateBuffer,
        randomWalkHitBuffer,visibilityHitBuffer, aperture, imgPlaneSize,
        forward, focalDistance, p1, right, up, spreadAngle, accumulatorOnePass, accumulator, constructLightBuffer,
        weightMeasureBuffer, probePixelIdx, screenParams,
        photomappingIdx, photomappingBuffer, camPos, constructEyeBuffer,
        eyePathBuffer,lightPathBuffer);
}

// EOF