#include "hip/hip_runtime.h"
/* .cuda.cu - Copyright 2019 Utrecht University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include ".cuda.h"

namespace lh2core {

// path tracing buffers and global variables
__constant__ CoreInstanceDesc* instanceDescriptors;
__constant__ CoreMaterial* materials;
__constant__ CoreLightTri* areaLights;
__constant__ CorePointLight* pointLights;
__constant__ CoreSpotLight* spotLights;
__constant__ CoreDirectionalLight* directionalLights;
__constant__ int4 lightCounts; // area, point, spot, directional
__constant__ uint* argb32;
__constant__ float4* argb128;
__constant__ uint* nrm32;
__constant__ float3* skyPixels;
__constant__ int skywidth;
__constant__ int skyheight;
__constant__ PathState* pathStates;
__constant__ float4* debugData;

// path tracer settings
__constant__ __device__ float geometryEpsilon;
__constant__ __device__ float clampValue;

// access
__host__ void SetInstanceDescriptors( CoreInstanceDesc* p ) { hipMemcpyToSymbol(HIP_SYMBOL( instanceDescriptors), &p, sizeof( void* ) ); }
__host__ void SetMaterialList( CoreMaterial* p ) { hipMemcpyToSymbol(HIP_SYMBOL( materials), &p, sizeof( void* ) ); }
__host__ void SetAreaLights( CoreLightTri* p ) { hipMemcpyToSymbol(HIP_SYMBOL( areaLights), &p, sizeof( void* ) ); }
__host__ void SetPointLights( CorePointLight* p ) { hipMemcpyToSymbol(HIP_SYMBOL( pointLights), &p, sizeof( void* ) ); }
__host__ void SetSpotLights( CoreSpotLight* p ) { hipMemcpyToSymbol(HIP_SYMBOL( spotLights), &p, sizeof( void* ) ); }
__host__ void SetDirectionalLights( CoreDirectionalLight* p ) { hipMemcpyToSymbol(HIP_SYMBOL( directionalLights), &p, sizeof( void* ) ); }
__host__ void SetLightCounts( int area, int point, int spot, int directional )
{
	const int4 counts = make_int4( area, point, spot, directional );
	hipMemcpyToSymbol(HIP_SYMBOL( lightCounts), &counts, sizeof( int4 ) );
}
__host__ void SetARGB32Pixels( uint* p ) { hipMemcpyToSymbol(HIP_SYMBOL( argb32), &p, sizeof( void* ) ); }
__host__ void SetARGB128Pixels( float4* p ) { hipMemcpyToSymbol(HIP_SYMBOL( argb128), &p, sizeof( void* ) ); }
__host__ void SetNRM32Pixels( uint* p ) { hipMemcpyToSymbol(HIP_SYMBOL( nrm32), &p, sizeof( void* ) ); }
__host__ void SetSkyPixels( float3* p ) { hipMemcpyToSymbol(HIP_SYMBOL( skyPixels), &p, sizeof( void* ) ); }
__host__ void SetSkySize( int w, int h ) { hipMemcpyToSymbol(HIP_SYMBOL( skywidth), &w, sizeof( int ) ); hipMemcpyToSymbol(HIP_SYMBOL( skyheight), &h, sizeof( int ) ); }
__host__ void SetPathStates( PathState* p ) { hipMemcpyToSymbol(HIP_SYMBOL( pathStates), &p, sizeof( void* ) ); }
__host__ void SetDebugData( float4* p ) { hipMemcpyToSymbol(HIP_SYMBOL( debugData), &p, sizeof( void* ) ); }

// access
__host__ void SetGeometryEpsilon( float e ) { hipMemcpyToSymbol(HIP_SYMBOL( geometryEpsilon), &e, sizeof( float ) ); }
__host__ void SetClampValue( float c ) { hipMemcpyToSymbol(HIP_SYMBOL( clampValue), &c, sizeof( float ) ); }

// counters for persistent threads
static __device__ Counters* counters;
__global__ void InitCountersForExtend_Kernel( int pathCount )
{
	if (threadIdx.x != 0) return;
	counters->activePaths = pathCount;	// remaining active paths
	counters->shaded = 0;				// persistent thread atomic for shade kernel
	counters->generated = 0;			// persistent thread atomic for generate in .optix.cu
	counters->extensionRays = 0;		// compaction counter for extension rays
	counters->shadowRays = 0;			// compaction counter for connections
	counters->connected = 0;
	counters->totalExtensionRays = pathCount;
	counters->totalShadowRays = 0;
}
__host__ void InitCountersForExtend( int pathCount ) { InitCountersForExtend_Kernel << <1, 32 >> > (pathCount); }
__global__ void InitCountersSubsequent_Kernel()
{
	if (threadIdx.x != 0) return;
	counters->totalExtensionRays += counters->extensionRays;
	counters->activePaths = counters->extensionRays;	// remaining active paths
	counters->extended = 0;				// persistent thread atomic for genSecond in .optix.cu
	counters->shaded = 0;				// persistent thread atomic for shade kernel
	counters->extensionRays = 0;		// compaction counter for extension rays
}
__host__ void InitCountersSubsequent() { InitCountersSubsequent_Kernel << <1, 32 >> > (); }
__host__ void SetCounters( Counters* p ) { hipMemcpyToSymbol(HIP_SYMBOL( counters), &p, sizeof( void* ) ); }

// functional blocks
#include "tools_shared.h"
#include "sampling_shared.h"
#include "material_shared.h"
#include "lights_shared.h"
#include "bsdf.h"
#include "pathtracer.h"
#include "finalize_shared.h"

} // namespace lh2core

// EOF