#include "hip/hip_runtime.h"
/* .optix.cu - Copyright 2019 Utrecht University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

   This file contains a minimal set of Optix functions. From here we will
   dispatch program flow to our own functions that implement the path tracer.
*/

#include "../kernels/noerrors.h"
#include "hip/hip_vector_types.h"

// global include files
#include "../../RenderSystem/common_settings.h"
#include "../../RenderSystem/common_types.h"
#include "../core_settings.h"

// global path tracing parameters
extern "C" { __constant__ Params params; }

// tools
__device__ __inline__ uint WangHash( uint s ) { s = (s ^ 61) ^ (s >> 16), s *= 9, s = s ^ (s >> 4), s *= 0x27d4eb2d, s = s ^ (s >> 15); return s; }
__device__ __inline__ uint RandomInt( uint& s ) { s ^= s << 13, s ^= s >> 17, s ^= s << 5; return s; }
__device__ __inline__ float RandomFloat( uint& s ) { return RandomInt( s ) * 2.3283064365387e-10f; }

static __inline __device__ float blueNoiseSampler( int x, int y, int sampleIdx, int sampleDimension )
{
	// wrap arguments
	x &= 127, y &= 127, sampleIdx &= 255, sampleDimension &= 255;

	// xor index based on optimized ranking
	int rankedSampleIndex = sampleIdx ^ params.blueNoise[sampleDimension + (x + y * 128) * 8 + 65536 * 3];

	// fetch value in sequence
	int value = params.blueNoise[sampleDimension + rankedSampleIndex * 256];

	// if the dimension is optimized, xor sequence value based on optimized scrambling
	value ^= params.blueNoise[(sampleDimension & 7) + (x + y * 128) * 8 + 65536];

	// convert to float and return
	return (0.5f + value) * (1.0f / 256.0f);
}

static __inline __device__ float3 RandomPointOnLens( const float r0, float r1 )
{
	const float blade = (int)(r0 * 9);
	float r2 = (r0 - blade * (1.0f / 9.0f)) * 9.0f;
	float x1, y1, x2, y2;
	__sincosf( blade * PI / 4.5f, &x1, &y1 );
	__sincosf( (blade + 1.0f) * PI / 4.5f, &x2, &y2 );
	if ((r1 + r2) > 1) r1 = 1.0f - r1, r2 = 1.0f - r2;
	const float xr = x1 * r1 + x2 * r2;
	const float yr = y1 * r1 + y2 * r2;
	float4 posLens = params.posLensSize;
	return make_float3( posLens ) + posLens.w * (params.right * xr + params.up * yr);
}

static __inline __device__ void generateEyeRay( float3& O, float3& D, const uint pixelIdx, const uint sampleIdx, uint& seed )
{
	// random point on pixel and lens
	int sx = pixelIdx % params.scrsize.x;
	int sy = pixelIdx / params.scrsize.x;
	float r0, r1, r2, r3;
	if (sampleIdx < 256)
		r0 = blueNoiseSampler( sx, sy, sampleIdx, 0 ),
		r1 = blueNoiseSampler( sx, sy, sampleIdx, 1 ),
		r2 = blueNoiseSampler( sx, sy, sampleIdx, 2 ),
		r3 = blueNoiseSampler( sx, sy, sampleIdx, 3 );
	else
		r0 = RandomFloat( seed ), r1 = RandomFloat( seed ),
		r2 = RandomFloat( seed ), r3 = RandomFloat( seed );
	O = RandomPointOnLens( r2, r3 );
	const float u = ((float)sx + r0) * (1.0f / params.scrsize.x);
	const float v = ((float)sy + r1) * (1.0f / params.scrsize.y);
	const float3 pointOnPixel = params.p1 + u * params.right + v * params.up;
	D = normalize( pointOnPixel - O );
}

#if __CUDA_ARCH__ >= 700
#define THREADMASK	__activemask() // volta, turing
#else
#define THREADMASK	0xffffffff // pascal, kepler, fermi
#endif

__device__ void setupPrimaryRay( const uint pathIdx, const uint stride )
{
	const uint pixelIdx = pathIdx % (params.scrsize.x * params.scrsize.y);
	const uint sampleIdx = pathIdx / (params.scrsize.x * params.scrsize.y) + params.pass;
	uint seed = WangHash( pathIdx * 16789 + params.pass * 1791 );
	// generate eye ray
	float3 O, D;
	generateEyeRay( O, D, pixelIdx, sampleIdx, seed );
	// populate path state array
	params.pathStates[pathIdx] = make_float4( O, __uint_as_float( (pathIdx << 8) + 1 /* S_SPECULAR in CUDA code */ ) );
	params.pathStates[pathIdx + stride] = make_float4( D, 0 );
	// trace eye ray
	uint u0, u1 = 0, u2 = 0xffffffff, u3 = __float_as_uint( 1e34f );
	optixTrace( params.bvhRoot, O, D, params.geometryEpsilon, 1e34f, 0.0f /* ray time */, OptixVisibilityMask( 1 ),
		OPTIX_RAY_FLAG_NONE, 0, 2, 0, u0, u1, u2, u3 );
	params.hitData[pathIdx] = make_float4( __uint_as_float( u0 ), __uint_as_float( u1 ), __uint_as_float( u2 ), __uint_as_float( u3 ) );
}

__device__ void setupSecondaryRay( const uint rayIdx, const uint stride )
{
	const float4 O4 = params.pathStates[rayIdx];
	const float4 D4 = params.pathStates[rayIdx + stride];
	float4 result = make_float4( 0, 0, __int_as_float( -1 ), 0 );
	uint pixelIdx = __float_as_uint( O4.w ) >> 8;
	uint u0, u1 = 0, u2 = 0xffffffff, u3 = __float_as_uint( 1e34f );
	optixTrace( params.bvhRoot, make_float3( O4 ), make_float3( D4 ), params.geometryEpsilon, 1e34f, 0.0f /* ray time */, OptixVisibilityMask( 1 ),
		OPTIX_RAY_FLAG_NONE, 0, 2, 0, u0, u1, u2, u3 );
	params.hitData[rayIdx] = make_float4( __uint_as_float( u0 ), __uint_as_float( u1 ), __uint_as_float( u2 ), __uint_as_float( u3 ) );
}

__device__ void generateShadowRay( const uint rayIdx, const uint stride )
{
	const float4 O4 = params.connectData[rayIdx]; // O4
	const float4 D4 = params.connectData[rayIdx + stride * MAXPATHLENGTH]; // D4
	// launch shadow ray
	uint u0 = 1;
	optixTrace( params.bvhRoot, make_float3( O4 ), make_float3( D4 ), params.geometryEpsilon, D4.w, 0.0f /* ray time */, OptixVisibilityMask( 1 ),
		OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT, 1, 2, 1, u0 );
	if (u0) return;
	const float4 E4 = params.connectData[rayIdx + stride * 2 * MAXPATHLENGTH]; // E4
	const int pixelIdx = __float_as_int( E4.w );
	params.accumulator[pixelIdx] += make_float4( E4.x, E4.y, E4.z, 1 );
}

extern "C" __global__ void __raygen__rg()
{
	const uint stride = params.scrsize.x * params.scrsize.y * params.scrsize.z;
	const uint3 idx = optixGetLaunchIndex();
	if (params.phase == 0)
	{
		// primary rays
		setupPrimaryRay( idx.x + idx.y * params.scrsize.x, stride );
	}
	else if (params.phase == 1)
	{
		// secondary rays
		setupSecondaryRay( idx.x + idx.y * params.scrsize.x, stride );
	}
	else
	{
		// shadow rays
		generateShadowRay( idx.x + idx.y * params.scrsize.x, stride );
	}
}

extern "C" __global__ void __miss__occlusion()
{
	optixSetPayload_0( 0u ); // instead of any hit. suggested by WillUsher.io.
}

extern "C" __global__ void __closesthit__radiance()
{
	const uint prim_idx = optixGetPrimitiveIndex();
	const uint inst_idx = optixGetInstanceIndex();
	const float2 bary = optixGetTriangleBarycentrics();
	const float tmin = optixGetRayTmax();
	optixSetPayload_0( (uint)(65535.0f * bary.x) + ((uint)(65535.0f * bary.y) << 16) );
	optixSetPayload_1( inst_idx );
	optixSetPayload_2( prim_idx );
	optixSetPayload_3( __float_as_uint( tmin ) );
}

// EOF